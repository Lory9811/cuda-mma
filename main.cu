#include <stdio.h>
#include <stdlib.h>

#include <iostream>
#include <vector>
#include <random>
#include <string>

#include <hip/hip_runtime.h>
#include <mma.h>




using namespace nvcuda;

// mm with tensor cores
__global__ void wmma16x16(half* a, half* b, half* c) {
	wmma::fragment<wmma::matrix_a, 16, 16, 16, half, wmma::row_major> a_frag;
	wmma::fragment<wmma::matrix_b, 16, 16, 16, half, wmma::row_major> b_frag;
	wmma::fragment<wmma::accumulator, 16, 16, 16, half> acc_frag;
	
	wmma::load_matrix_sync(a_frag, a, 16);
	wmma::load_matrix_sync(b_frag, b, 16);

	wmma::mma_sync(acc_frag, a_frag, b_frag, acc_frag);

	wmma::store_matrix_sync(c, acc_frag, 16, wmma::mem_row_major);
}

// mm without tensor cores
__global__ void mma16x16(half* a, half* b, half* c) {
	int i = blockIdx.x * blockDim.x + threadIdx.x;
	int j = blockIdx.y * blockDim.y + threadIdx.y;

	if (i >= 16 || j >= 16) {
		return;
	}

	half sum = c[i + j * 16];
	for (int k = 0; k < 16; k++) {
		sum += a[j * 16 + k] * b[i + k * 16];
	}

	c[i + j * 16] = sum;
}

std::vector<half> randomMatrix(std::uniform_real_distribution<float> distribution, int size) {
	std::random_device rd;
	std::default_random_engine engine(rd());
	std::vector<half> matrix;
	
	for (int i = 0; i != size; i++) {
		matrix.emplace_back(__float2half(distribution(engine)));
	}

	return matrix;
}

void checkCudaError(hipError_t error, int line) {
	if (error != hipSuccess) {
		std::cout << "(" << line << ") Cuda error: " << hipGetErrorString(error) << std::endl;
	}
}

#define checkCudaError(x) checkCudaError(x, __LINE__)

std::vector<half> runWmmaDemo(std::vector<half> a, std::vector<half> b, float* time) {
	std::vector<half> c(16 * 16, 10.0);

	half* device_a;
	half* device_b;
	half* device_c;

	checkCudaError(hipMallocManaged(&device_a, sizeof(half) * 16 * 16));
	checkCudaError(hipMallocManaged(&device_b, sizeof(half) * 16 * 16));
	checkCudaError(hipMallocManaged(&device_c, sizeof(half) * 16 * 16));

	checkCudaError(hipMemcpy(device_a, a.data(), sizeof(half) * a.size(), hipMemcpyHostToDevice));
	checkCudaError(hipMemcpy(device_b, b.data(), sizeof(half) * b.size(), hipMemcpyHostToDevice));

	hipEvent_t start, stop;
	checkCudaError(hipEventCreate(&start));
	checkCudaError(hipEventCreate(&stop));

	checkCudaError(hipEventRecord(start));				// Start performance measurement

	// TODO: set proper block/grid size
	dim3 threadsPerBlock(16, 16, 1);
	dim3 numBlocks(1);
	wmma16x16<<<numBlocks, threadsPerBlock>>>(device_a, device_b, device_c);
	hipError_t errSync = hipGetLastError();
	hipError_t errAsync = hipDeviceSynchronize();
	if (errSync != hipSuccess) {
		printf("Sync kernel error: %s\n", hipGetErrorString(errSync));
	}
	if (errAsync != hipSuccess) {
		printf("Async kernel error: %s\n", hipGetErrorString(errAsync));
	}

	checkCudaError(hipEventRecord(stop));				// Stop measurement
	checkCudaError(hipEventSynchronize(stop));
		
	checkCudaError(hipEventElapsedTime(time, start, stop));

	checkCudaError(hipMemcpy(c.data(), device_c, sizeof(half) * c.size(), hipMemcpyDeviceToHost));

	checkCudaError(hipFree(device_a));
	checkCudaError(hipFree(device_b));
	checkCudaError(hipFree(device_c));

	return c;
}

std::vector<half> runMmaDemo(std::vector<half> a, std::vector<half> b, float* time) {
	std::vector<half> c(16 * 16, 10.0);

	half* device_a;
	half* device_b;
	half* device_c;

	checkCudaError(hipMallocManaged(&device_a, sizeof(half) * 16 * 16));
	checkCudaError(hipMallocManaged(&device_b, sizeof(half) * 16 * 16));
	checkCudaError(hipMallocManaged(&device_c, sizeof(half) * 16 * 16));

	checkCudaError(hipMemcpy(device_a, a.data(), sizeof(half) * a.size(), hipMemcpyHostToDevice));
	checkCudaError(hipMemcpy(device_b, b.data(), sizeof(half) * b.size(), hipMemcpyHostToDevice));

	dim3 threadsPerBlock(16, 16, 1);
	dim3 numBlocks(1);
	hipEvent_t start, stop;
	checkCudaError(hipEventCreate(&start));
	checkCudaError(hipEventCreate(&stop));

	checkCudaError(hipEventRecord(start));				// Start performance measurement

	mma16x16<<<numBlocks, threadsPerBlock>>>(device_a, device_b, device_c);

	hipError_t errSync = hipGetLastError();
	hipError_t errAsync = hipDeviceSynchronize();
	if (errSync != hipSuccess) {
		printf("Sync kernel error: %s\n", hipGetErrorString(errSync));
	}
	if (errAsync != hipSuccess) {
		printf("Async kernel error: %s\n", hipGetErrorString(errAsync));
	}

	checkCudaError(hipEventRecord(stop));				// Stop measurement
	checkCudaError(hipEventSynchronize(stop));
		
	checkCudaError(hipEventElapsedTime(time, start, stop));

	checkCudaError(hipMemcpy(c.data(), device_c, sizeof(half) * c.size(), hipMemcpyDeviceToHost));

	checkCudaError(hipFree(device_a));
	checkCudaError(hipFree(device_b));
	checkCudaError(hipFree(device_c));

	return c;
}

int main() {
	std::uniform_real_distribution<float> distribution(-10, 10);
	
	std::vector<half> a = randomMatrix(distribution, 16 * 16);
	std::vector<half> b = randomMatrix(distribution, 16 * 16);

	float time_wmm = 0.0f;
	float time_mm = 0.0f;
	std::vector<half> c_wmm = runWmmaDemo(a, b, &time_wmm);
	std::vector<half> c_mm = runMmaDemo(a, b, &time_mm);
	
	std::cout << "With Tensor Cores: " <<  std::to_string(time_wmm) << " ms" << std::endl;
	std::cout << "Cuda: " <<  std::to_string(time_mm) << " ms" << std::endl;

	int mismatches = 0;
	for (int i = 0; i < 16 * 16; i++) {
		if (abs(__half2float(c_wmm[i]) - __half2float(c_mm[i])) > 0.5) {
			std::cout << __half2float(c_wmm[i]) << " " << __half2float(c_mm[i]) << std::endl;
			mismatches += 1;
		}
	}

	if (mismatches > 0) {
		std::cout << mismatches << " wrong" << std::endl;
	}

	return 0;
}
